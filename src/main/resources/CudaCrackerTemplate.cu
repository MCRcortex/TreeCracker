#include "hip/hip_runtime.h"
#include <thread>
#include <vector>
#include <iostream>
#include <string>
#include <fstream>
#include <stdlib.h>
#include <utility>
#include <mutex>
#include <map>
#include <algorithm>
#define _USE_MATH_DEFINES
#include <math.h>
#include <chrono>


uint64_t millis() {return (std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch())).count();}

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
	fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
	exit(code);
  }
}

#define SETGPU(gpuId) hipSetDevice(gpuId);\
	GPU_ASSERT(hipPeekAtLastError());\
	GPU_ASSERT(hipDeviceSynchronize());\
	GPU_ASSERT(hipPeekAtLastError());

#define DEVICEABLE __host__ __device__



#define THREAD_SIZE 256LLU
#define BLOCK_SIZE (1LLU<<28) //(1LLU<<29)
#define BATCH_SIZE (THREAD_SIZE * BLOCK_SIZE)



__managed__ uint32_t count = 0;
__managed__ uint64_t seedBuff[60000000];


__managed__ uint32_t countOut = 0;
__managed__ uint64_t outputSeedBuff[6000];//Max seed output for the secondary filter


__global__ __launch_bounds__(THREAD_SIZE) void InitalFilter(const uint64_t offset) {
	uint64_t seed = (((uint64_t)blockIdx.x * (uint64_t)blockDim.x + (uint64_t)threadIdx.x)) + offset;

	PRIMARY_TREE_FILTER

	//TODO: Have different seed buffers per thread or somthing, so that the atomicAdd isnt a bottleneck
	seedBuff[atomicAdd(&count, 1)] = (((uint64_t)blockIdx.x * (uint64_t)blockDim.x + (uint64_t)threadIdx.x)) + offset;
}





AUX_TREE_FUNCTIONS_REPLACEMENT



#define NEXT_INT_16(seed) (((seed = ((seed * 0x5DEECE66DLLU + 0xBLLU)&((1LLU<<48)-1)))>>(48-4)))

#define TREE_TEST(testMethod, index, expected_x, expected_z, IF_TYPE) IF_TYPE ((!(mask & (1<<index))) && x_pos == expected_x && z_pos == expected_z) mask |= ((uint8_t)testMethod(seed))<<index;
#define TARGET_MASK ((1<<AUXILIARY_TREE_COUNT)-1)
__global__ __launch_bounds__(THREAD_SIZE) void SecondaryFilter() {
	uint64_t idx = ((((uint64_t)blockIdx.x * (uint64_t)blockDim.x + (uint64_t)threadIdx.x)));
	if (idx >= count)
		return;
	uint64_t seed = seedBuff[idx];
	seed = LCG_REVERSE_STAGE_2_REPLACEMENT;
	
	uint8_t mask = 0;
	int32_t x_pos;
	int32_t z_pos = NEXT_INT_16(seed);
	for (int32_t index = 0; index < MAX_TREE_RNG_RANGE_REPLACEMENT * 2 && mask != TARGET_MASK; index++) {
        x_pos = z_pos;
		z_pos = NEXT_INT_16(seed);
		
        AUX_TREE_TEST_INNER_LOOP_CALL_REPLACEMENT
		

	}
	
	
	if (mask != TARGET_MASK)
		return;
	
	outputSeedBuff[atomicAdd(&countOut, 1)] = seedBuff[idx];
}











int main() {
	SETGPU(0);
	std::ofstream outfile("output_seeds.dat", std::ofstream::binary);
	for (uint64_t offset = 0; offset < (1LLU<<44); offset += BATCH_SIZE) {
		uint64_t start = millis();
		
		count = 0;
		countOut = 0;
		InitalFilter<<<BLOCK_SIZE, THREAD_SIZE>>>((((uint64_t)INIT_TREE_INNER_X) << 44) | offset);
		GPU_ASSERT(hipPeekAtLastError());	
		GPU_ASSERT(hipDeviceSynchronize());
		GPU_ASSERT(hipPeekAtLastError());
		uint64_t step1 = millis()-start;
		start = millis();

        uint64_t step2 = 0;
        uint64_t step3 = 0;
		if (count != 0) {
            SecondaryFilter<<<ceil((double)count/THREAD_SIZE), THREAD_SIZE>>>();
            GPU_ASSERT(hipPeekAtLastError());
            GPU_ASSERT(hipDeviceSynchronize());
            GPU_ASSERT(hipPeekAtLastError());
            step2 = millis()-start;
            start = millis();

            for (uint64_t index = 0; index < countOut; index++) {
                outfile << outputSeedBuff[index] << std::endl;
                outfile.flush();
            }
            step3 = millis()-start;
		}

		std::cout << "Finished gpu: " << (step1+step2+step3) << ", " << step1 << ", " << step2 << ", " << step3 << ", " << count << ", " << countOut << ", " << (((1LLU<<44) - offset)/BATCH_SIZE)  << std::endl;
	}
	outfile.close();
	return 1;
}